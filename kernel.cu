#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include "GPUMath.h"
#include <iomanip>
#include <cstring>

#ifdef USE_SYMMETRY
#define KSIZE 11
#else
#define KSIZE 10
#endif
#define GPU_GRP_SIZE 128
#define NB_JUMP 32
#define NUM_GTABLE_CHUNK 16    // Number of GTable chunks that are pre-computed and stored in global memory
#define NUM_GTABLE_VALUE 65536 // Number of GTable values per chunk (all possible states) (2 ^ NUM_GTABLE_CHUNK)
#define SIZE_GTABLE_POINT 32   // Each Point in GTable consists of two 32-byte coordinates (X and Y)
#define COUNT_GTABLE_POINTS (NUM_GTABLE_CHUNK * NUM_GTABLE_VALUE)
#define COUNT_CUDA_THREADS (BLOCKS_PER_GRID * THREADS_PER_BLOCK)

__constant__ int CHUNK_FIRST_ELEMENT[NUM_GTABLE_CHUNK] = {
  65536 * 0,  65536 * 1,  65536 * 2,  65536 * 3,
  65536 * 4,  65536 * 5,  65536 * 6,  65536 * 7,
  65536 * 8,  65536 * 9,  65536 * 10, 65536 * 11,
  65536 * 12, 65536 * 13, 65536 * 14, 65536 * 15,
};

void loadArrayFromFile(const char* filename, uint8_t* array, size_t size) {
    FILE* file = fopen(filename, "rb");
    if (file == nullptr) {
        printf("Failed to open file %s for reading.\n", filename);
        return;
    }
    fread(array, sizeof(uint8_t), size, file);
    fclose(file);
}

void convertPrivateKey(const char* privKeyStr, uint16_t* privKey) {
    for (int i = 0; i < 16; ++i) {
        uint16_t value = 0;
        sscanf(&privKeyStr[i * 4], "%4hx", &value);
        privKey[i] = value;
    }
}

void hexStringToBytes(const char* hexString, uint8_t* output) {
    size_t len = strlen(hexString);
    for (size_t i = 0; i < len / 2; ++i) {
        sscanf(hexString + 2 * i, "%2hhx", &output[i]);
    }
}

__device__ void __forceinline__  _PointMultiSecp256k1(uint64_t* qx, uint64_t* qy, uint16_t* privKey, const uint8_t* __restrict__ gTableX, const uint8_t* __restrict__ gTableY) {

    int chunk = 0;
    uint64_t qz[5] = { 1, 0, 0, 0, 0 };

    //Find the first non-zero point [qx,qy]
    for (; chunk < NUM_GTABLE_CHUNK; chunk++) {
        if (privKey[chunk] > 0) {
            int index = (CHUNK_FIRST_ELEMENT[chunk] + (privKey[chunk] - 1)) * SIZE_GTABLE_POINT;
            memcpy(qx, gTableX + index, SIZE_GTABLE_POINT);
            memcpy(qy, gTableY + index, SIZE_GTABLE_POINT);
            chunk++;
            break;
        }
    }

    //Add the remaining chunks together
    for (; chunk < NUM_GTABLE_CHUNK; chunk++) {
        if (privKey[chunk] > 0) {
            uint64_t gx[4]{};
            uint64_t gy[4]{};

            int index = (CHUNK_FIRST_ELEMENT[chunk] + (privKey[chunk] - 1)) * SIZE_GTABLE_POINT;

            memcpy(gx, gTableX + index, SIZE_GTABLE_POINT);
            memcpy(gy, gTableY + index, SIZE_GTABLE_POINT);

            _PointAddSecp256k1(qx, qy, qz, gx, gy);
        }
    }

    //Performing modular inverse on qz to obtain the public key [qx,qy]
    _ModInv(qz);
    _ModMult(qx, qz);
    _ModMult(qy, qz);

}


__global__ void kernel_PointMultiSecp256k1(uint64_t* qx, uint64_t* qy, uint16_t* privKey, uint8_t* gTableX, uint8_t* gTableY) {
    _PointMultiSecp256k1(qx, qy, privKey, gTableX, gTableY);
}

int main() {
    uint8_t* gTableX = new uint8_t[COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT]{};
    uint8_t* gTableY = new uint8_t[COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT]{};

    int numElements = COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT;

    loadArrayFromFile("gTableX.dat", gTableX, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);
    loadArrayFromFile("gTableY.dat", gTableY, COUNT_GTABLE_POINTS * SIZE_GTABLE_POINT);

    uint8_t* d_x;
    uint8_t* d_y;

    hipMalloc((void**)&d_x, numElements * sizeof(uint8_t));
    hipMalloc((void**)&d_y, numElements * sizeof(uint8_t));

    hipMemcpyAsync(d_x, gTableX, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_y, gTableY, numElements * sizeof(uint8_t), hipMemcpyHostToDevice);

    /*
    Test PrivateKey: 6b86b273ff34fce19d6b804eff5a3f5747ada4eaa22f1d49c01e52ddb7875b4b

    Correct result:
    X: 114867258463794774232047586919768456742067559961833201576133440978330520935359
    Y: 90720385063194657765607368302607820582973541301350451862086334612521918372085
    OR
    Public Key: 03 fdf4907810a9f5d9462a1ae09feee5ab205d32798b0ffcc379442021f84c5bbf
    */

    const char* privKeyStr = "6b86b273ff34fce19d6b804eff5a3f5747ada4eaa22f1d49c01e52ddb7875b4b";
    uint16_t privKey[16];
    convertPrivateKey(privKeyStr, privKey);

    std::cout << "privKey (host): ";
    for (int i = 0; i < 16; i++) {
        std::cout << std::hex << privKey[i] << " ";
    }
    std::cout << std::endl;

    size_t keyLen = strlen(privKeyStr) / 2;
    uint8_t* privKeyBytes = new uint8_t[keyLen];
    hexStringToBytes(privKeyStr, privKeyBytes);
    std::cout << "Private key bytes: ";
    for (size_t i = 0; i < keyLen; ++i) {
        std::cout << (int)privKeyBytes[i] << " ";
    }
    std::cout << std::endl;



    uint16_t* privKeyGPU;
    hipMalloc((void**)&privKeyGPU, sizeof(uint16_t) * 16);
    hipMemcpy(privKeyGPU, privKeyBytes, sizeof(uint16_t) * 16, hipMemcpyHostToDevice);

    uint64_t* qxGPU, * qyGPU;
    hipMalloc((void**)&qxGPU, sizeof(uint64_t) * 4);
    hipMalloc((void**)&qyGPU, sizeof(uint64_t) * 4);

    kernel_PointMultiSecp256k1 << <1, 1 >> > (qxGPU, qyGPU, privKeyGPU, d_x, d_y);
    hipDeviceSynchronize();

    uint64_t qx[4], qy[4];
    hipMemcpy(qx, qxGPU, sizeof(uint64_t) * 4, hipMemcpyDeviceToHost);
    hipMemcpy(qy, qyGPU, sizeof(uint64_t) * 4, hipMemcpyDeviceToHost);


    std::cout << "Public key X: ";
    for (int i = 0; i < 4; i++) {
        std::cout << qx[i];
    }
    std::cout << std::endl;

    std::cout << "Public key Y: ";
    for (int i = 0; i < 4; i++) {
        std::cout << qy[i];
    }
    std::cout << std::endl;


    hipFree(d_x);
    hipFree(d_y);
    hipFree(privKeyGPU);
    hipFree(qxGPU);
    hipFree(qyGPU);
    delete[] gTableX;
    delete[] gTableY;
    return 0;
}




